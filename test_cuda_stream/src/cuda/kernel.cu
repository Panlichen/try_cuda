
#include <hip/hip_runtime.h>
#define N (1024*1024)  
#define FULL_DATA_SIZE N*20  

extern "C"
__global__ void kernel(int* a, int *b, int*c)
{
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
 
	if (threadID < N)
	{
		c[threadID] = (a[threadID] + b[threadID]) / 2;
	}
}